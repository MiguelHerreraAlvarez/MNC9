#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <ctime>
#include <stdio.h>
#include <cstdio>
#include <cstdlib>

#define tam_blocks 2
hipError_t addWithCuda(double* c, const double* a, const double* b, unsigned int size);

__global__ void addKernel(double* c, const double* a, const double* b, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    float sum = 0;
    for (int k = 0; k < N; k++)
        sum += a[i * N + k] * b[k * N + j];
    c[i * N + j] = sum;
}

int main()
{
    for (int k = 64; k <= 1024; k = k * 2) {
        const int arraySize = k;
        double* A = (double*)malloc(sizeof(double) * arraySize * arraySize);
        double* B = (double*)malloc(sizeof(double) * arraySize * arraySize);
        double* C = (double*)malloc(sizeof(double) * arraySize * arraySize);

        for (int i = 0; i < arraySize; i++)
        {
            for (int j = 0; j < arraySize; j++) {
                A[arraySize * i + j] = i + j;
                B[arraySize * i + j] = i - j;
            }
        }
        hipError_t cudaStatus;
        // Add vectors in parallel.
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        for (int i = 0; i < 10; i++) {
            cudaStatus = addWithCuda(C, A, B, arraySize);
        }
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addWithCuda failed!");
            return 1;
        }
        printf("Tiempo CUDA: %f milisegundos, N = %d\n", ((float)milliseconds / 10.) / CLOCKS_PER_SEC, arraySize);

        /*if (arraySize == 64) {
            printf("Diagonal de la matriz de tama�o 64*64");
            for (int i = 0; i < arraySize; i++)
            {
                printf(" %lf ", C[arraySize * i + i]);
            }
        }*/
        // hipDeviceReset must be called before exiting in order for profiling and
        // tracing tools such as Nsight and Visual Profiler to show complete traces.
        cudaStatus = hipDeviceReset();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceReset failed!");
            return 1;
        }

    }
    
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(double* c, const double* a, const double* b, unsigned int size)
{
    double* dev_a = 0;
    double* dev_b = 0;
    double* dev_c = 0;
    int total_size = size * size;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, total_size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, total_size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, total_size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, total_size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, total_size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    if (size > 32) {
        dim3 threadsPerBlock(size / 32, size / 32);
        dim3 numBlocks(tam_blocks, tam_blocks);
        // Launch a kernel on the GPU with one thread for each element.
        addKernel << <numBlocks, threadsPerBlock >> > (dev_c, dev_a, dev_b, size);
    }
    else {
        dim3 threadsPerBlock(size, size);
        dim3 numBlocks(tam_blocks, tam_blocks);
        // Launch a kernel on the GPU with one thread for each element.
        addKernel << <numBlocks, threadsPerBlock >> > (dev_c, dev_a, dev_b, size);
    }
    
    
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, total_size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}

