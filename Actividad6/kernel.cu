#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>

hipError_t matMulWithCuda(double* c, const double* a, const double* b, unsigned int size);

__global__ void matMulKernel(double* c, const double* a, const double* b, int N)
{
	int i = threadIdx.x;
	int j = threadIdx.y;
	double sum = 0;
	for (int k = 0; k < N; k++) {
		sum += a[i * N + k] * b[k * N + j];
	}
	c[i * N + j] = sum;
}

int main()
{
	for (int N = 1; N <= 32; N = N * 2) {
		printf("SIZE = %d \n", N);
		double* c = (double*)malloc(sizeof(double) * N * N);
		double* a = (double*)malloc(sizeof(double) * N * N);
		double* b = (double*)malloc(sizeof(double) * N * N);
		for (int i = 0; i < N; i++) {
			for (int j = 0; j < N; j++) {
				a[i * N + j] = (float)i + j;
				b[i * N + j] = (float)i - j;
			}
		}
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipError_t cudaStatus = matMulWithCuda(c, a, b, N);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addWithCuda failed!");
			return 1;
		}

		/*for (int i = 0; i < N * N; i++) {
			if (i % N == 0) printf("\n");
			printf("%lf ", c[i]);
		}*/

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}

		free(a);
		free(b);
		free(c);
	}
	return 0;
}

hipError_t matMulWithCuda(double* c, const double* a, const double* b, unsigned int ldim)
{
	double* dev_a = 0;
	double* dev_b = 0;
	double* dev_c = 0;
	hipError_t cudaStatus;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int matrixDim = ldim * ldim;
	hipEventRecord(start);
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, matrixDim * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, matrixDim * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, matrixDim * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, matrixDim * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, matrixDim * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	dim3 threadsPerBlock(ldim, ldim);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("\tCOPY TIME: %lfms\n", milliseconds);
	// Launch a kernel on the GPU with one thread for each element.
	hipEventRecord(start);
	for (int count = 0; count < 10; count++) {
		matMulKernel << <1, threadsPerBlock >> > (dev_c, dev_a, dev_b, ldim);
	}
	hipEventRecord(stop);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, matrixDim * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	hipEventSynchronize(stop);
	milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("\tCORE TIME: %lfms\n", milliseconds / 10);
Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}


