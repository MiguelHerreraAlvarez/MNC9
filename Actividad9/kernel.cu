#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

hipError_t matMulWithCuda(double* c, const double* a, const double* b, unsigned int size);

__global__ void matMulKernel(double* c, const double* a, const double* b, int N)
{
	double sum = 0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	for (int k = 0; k < N; k++) {
		sum += a[row * N + k] * b[k * N + col];
	}
	c[row * N + col] = sum;
}

int main()
{
	int N = 64;
	for (int N = 64; N <= 1024; N = N * 2) {
		printf("SIZE = %d\t", N);

		double* a = (double*)malloc(sizeof(double) * N * N);
		double* b = (double*)malloc(sizeof(double) * N * N);
		double* c = (double*)malloc(sizeof(double) * N * N);
		
		for (int i = 0; i < N; i++) {
			for (int j = 0; j < N; j++) {
				a[i * N + j] = (float)i + j;
				b[i * N + j] = (float)i - j;
			}
		}
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		// Add vectors in parallel.
		hipError_t cudaStatus = matMulWithCuda(c, a, b, N);
		//hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addWithCuda failed!");
			return 1;
		}

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}

		free(a);
		free(b);
		free(c);
	}
	return 0;
}

hipError_t matMulWithCuda(double* c, const double* a, const double* b, unsigned int ldim)
{

	hipError_t cudaStatus;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	for (int i = 0; i <= 10; i++) {
		double* dev_a = 0;
		double* dev_b = 0;
		double* dev_c = 0;
		//if (ldim <= 32) 
		int matrixDim = ldim * ldim;
		//else matrixDim = 32 * 32;

		// Choose which GPU to run on, change this on a multi-GPU system.
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}

		// Allocate GPU buffers for three vectors (two input, one output)    .
		cudaStatus = hipMalloc((void**)&dev_c, matrixDim * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_a, matrixDim * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_b, matrixDim * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_a, a, matrixDim * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_b, b, matrixDim * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		int dimension = 32;
		//printf("Calculo para %d = %d", ldim, ldim / dimension);

		dim3 dimGrid(ldim / dimension, ldim / dimension);
		//int dimGrid = ldim / 32;

		float milliseconds = 0;
		// Launch a kernel on the GPU with one thread for each element.


		dim3 threadsPerBlock(dimension, dimension);
		//for (int count = 0; count < 10; count++) {
		matMulKernel << <dimGrid, threadsPerBlock >> > (dev_c, dev_a, dev_b, ldim);
		//}

		hipEventRecord(stop);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(c, dev_c, matrixDim * sizeof(double), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

	Error:
		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);
	}
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("TIME: %lfms\n", milliseconds / 10);
	return cudaStatus;
}


